#include "hip/hip_runtime.h"
#include "kernel.h"
#define TPB 64
#define RAD 1

__global__ void ddKernel(float *d_out, float *d_in, int size, float h){
    const int idx = threadIdx.x + blockDim.x*blockIdx.x;
    if(idx>size) return;



    extern __shared__ float s_in[];
    const int s_idx = threadIdx.x + RAD;


    s_in[s_idx] = d_in[idx];
    if(threadIdx.x<RAD){
        s_in[s_idx-RAD] = d_in[idx-RAD];
        s_in[s_idx+blockDim.x] = d_in[idx+blockDim.x];
    }

    __syncthreads();
    d_out[idx] = (s_in[s_idx-1]+s_in[s_idx+1] - 2.f*s_in[s_idx])/(h*h);
}

void ddParallel(float *out, const float *in, int n, float h){
    float *d_in = 0, *d_out = 0;
    hipMalloc(&d_in, n*sizeof(float));
    hipMalloc(&d_out, n*sizeof(float));

    hipMemcpy(d_in, in, n*sizeof(float), hipMemcpyHostToDevice);

    const size_t smemSize = (TPB + 2*RAD)*sizeof(float);

    ddKernel<<<(n+TPB-1)/TPB, TPB, smemSize>>>(d_out, d_in, n, h);

    hipMemcpy(out, d_out, n*sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_out);
    hipFree(d_in);
    
}
