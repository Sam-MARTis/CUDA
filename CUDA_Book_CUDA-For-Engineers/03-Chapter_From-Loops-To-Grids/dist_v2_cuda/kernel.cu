
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 64
#define TPB 32

__device__ float distance(float x1, float x2){
    return sqrt((x2-x1)*(x2-x1));
}

__global__ void distanceKernel(float *d_out, float *d_in, float ref, int len){
    const int i = blockIdx.x*blockDim.x + threadIdx.x;
    const float x = d_in[i];
    d_out[i] = distance(x, ref);
    printf(" = %2d: dist from %f to %f is %f.\n", i, ref, x, d_out[i]);
}

__host__ void distanceArray(float *out, float *in, float ref, int len){
    float *d_in = 0;
    float *d_out = 0;

    hipMalloc(&d_in,  len*sizeof(float));
    hipMalloc(&d_out, len*sizeof(float));
    
    hipMemcpy(d_in, in, len*sizeof(float), hipMemcpyHostToDevice);

    distanceKernel<<<N/TPB, TPB>>>(d_out, d_in, ref, N);

    hipMemcpy(out, d_out, len*sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_in);
    hipFree(d_out);
}